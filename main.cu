#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <math.h>
#include <hiprand/hiprand_kernel.h>

#include "shapes.h"
#include "vec_math.h"
#include "bsdf.h"


#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";
        hipDeviceReset();
        exit(99);
    }
}


typedef struct config_s {
    vec3 camera_pos;
    int max_x;
    int max_y;
    float focal_length;
    float view_port_height;
} config_t;

__global__ void render(vec3 *fb, config_t config, Shape** scene, int n_objects) {
    int max_x = config.max_x;
    int max_y = config.max_y;
    
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x || j >= max_y))
        return;
    int pixel_index = j*max_x + i;

    vec3 camera_pos = config.camera_pos;
    float focal_length = config.focal_length;
    float viewport_height = config.view_port_height;
    float viewport_width = viewport_height * (static_cast<float>(max_x)/max_y);
    
    vec3 viewport_u = vec3(viewport_width, 0, 0);
    vec3 viewport_v = vec3(0, -viewport_height, 0);

    vec3 pixel_delta_u = viewport_u / (float) max_x;
    vec3 pixel_delta_v = viewport_v / (float) max_y;

    vec3 viewport_upper_left = camera_pos - vec3(0, 0, focal_length) - viewport_u/2.0 - viewport_v/2.0;
    vec3 pixel00_loc = viewport_upper_left + (pixel_delta_u + pixel_delta_v) * 0.5;

    int n_samples = 1000;
    vec3 p_color(0, 0, 0);
    vec3 totalLight(0, 0, 0);
    int n_origin;
    hiprandState s;

    hiprand_init(pixel_index, 0, 0, &s);
    for (int sample = 0; sample < n_samples; sample++) {
        vec3 p_center = pixel00_loc + (pixel_delta_u * (i - 0.5f + hiprand_uniform(&s))) + (pixel_delta_v * (max_y - j - 0.5f + hiprand_uniform(&s)));
        vec3 direction = (p_center - camera_pos).normalized();
        ray r = ray(camera_pos, direction);
        totalLight += tracePath(scene, n_objects, r, &s);
    }
    p_color = totalLight/((float)n_samples);
    fb[pixel_index] = p_color;
}

__global__ void constructScene(Shape **scene) {
    if (threadIdx.x == 0 && blockIdx.x == 0){
        Eigen::Affine3f t1 = IDENTITY;
        t1.translation() = Eigen::Translation3f(0.5, 0, -3.0).translation();
        
        /* Floor */
        Eigen::Affine3f floor = IDENTITY;
        floor.linear() = Eigen::AngleAxisf(PI/2, vec3(1, 0, 0)).toRotationMatrix();
        floor.translation() = Eigen::Translation3f(0.0, -1.0, 0.0).translation();

        /* Back Wall */
        Eigen::Affine3f back = IDENTITY;
        back.translation() = Eigen::Translation3f(0.0, 0, -5.0).translation();

        /* Left Wall */
        Eigen::Affine3f left = IDENTITY;
        left.linear() = Eigen::AngleAxisf(PI/2, vec3(0, 1, 0)).toRotationMatrix();
        left.translation() = Eigen::Translation3f(-1.0, 0.0, -5.0).translation();
        
        /* Right Wall */
        Eigen::Affine3f right = IDENTITY;
        right.linear() = Eigen::AngleAxisf(-PI/2, vec3(0, 1, 0)).toRotationMatrix();
        right.translation() = Eigen::Translation3f(1.0, 0.0, -5.0).translation();
                
        /* Ceiling */
        Eigen::Affine3f ceiling = IDENTITY;
        ceiling.linear() = Eigen::AngleAxisf(-PI/2, vec3(1, 0, 0)).toRotationMatrix();
        ceiling.translation() = Eigen::Translation3f(0.0, 1.0, 0.0).translation();

        /* Front Wall */
        Eigen::Affine3f tplane6 = IDENTITY;
        tplane6.linear() = Eigen::AngleAxisf(PI, vec3(0, 1, 0)).toRotationMatrix();
        tplane6.translation() = Eigen::Translation3f(0.0, 0.0, 7.0).translation();

        Eigen::Affine3f t3 = IDENTITY;
        t3.translation() = Eigen::Translation3f(0.5, -0.5, -4.0).translation();

        Eigen::Affine3f t4 = IDENTITY;
        t4.translation() = Eigen::Translation3f(-0.5, -0.5, -3.5).translation();

        Eigen::Affine3f t5 = IDENTITY;
        t5.translation() = Eigen::Translation3f(0.0, -0.5, -2.5).translation();
        t5.linear() = Eigen::AngleAxisf(PI/4, vec3(1, 0, 0)).toRotationMatrix();

        Material light_material;
        light_material.emissive = vec3(1.0f, 0.9f, 0.7f);   

        Material base;
        base.albedo = vec3(0.4f, 0.4f, 0.4f);

        Material green;
        green.albedo = vec3(0.0f, 1.0f, 0.0f);

        Material red;
        red.albedo = vec3(1.0f, 0.0f, 0.0f);

        Material glass;  
        glass.f0 = 0.02f;
        glass.specularRoughness = 0;
        glass.specularColor = vec3(1.0f, 1.0f, 1.0f) * 0.8f;
        glass.IOR = 1.5f;
        glass.transparency = 1.0f;
        glass.refractionRoughness = 0.1f;

        Material metal;
        metal.albedo = vec3(1.0f, 1.0f, 1.0f);    
        metal.f0 = 1.0f;
        metal.specularRoughness = 0.05f;
        metal.specularColor = vec3(0.7f, 0.1f, 0.8f);   

        Material dielectric;
        dielectric.albedo = vec3(0.9f, 0.3f, 0.7f);
        dielectric.emissive = vec3(0.0f, 0.0f, 0.0f);        
        dielectric.f0 = 0.1f;
        dielectric.specularRoughness = 0.2f;
        dielectric.specularColor = vec3(0.9f, 0.9f, 0.9f);   

        scene[0] = new Plane(base, floor, -15, -15, 15, 15);
        scene[1] = new Plane(red, left, -15, -15, 15, 15);
        scene[2] = new Plane(green, right, -15, -15, 15, 15);
        scene[3] = new Plane(base, back, -15, -15, 15, 15);
        scene[4] = new Plane(light_material, ceiling, -15, -15, 15, 15);
        scene[5] = new Plane(base, tplane6, -15, -15, 15, 15);
        
        scene[6] = new Cube(dielectric, t3);
        scene[7] = new Cylinder(metal, t4);
        scene[8] = new Sphere(glass, t5);
        //scene[8] = new Cube(glass, t5);
    }
}

int main() {

    int nx = 800;
    int ny = 800;

    int tx = 8;
    int ty = 8;

    int num_pixels = nx*ny;

    size_t fb_size = num_pixels*sizeof(vec3);

    /* Allocate space for Scene*/
    int n_objs = 9;
    Shape **scene;

    checkCudaErrors(hipMalloc((void **)&scene, n_objs*sizeof(void**)));
    checkCudaErrors(hipDeviceSynchronize());

    constructScene<<<1, 1>>>(scene);
    checkCudaErrors(hipDeviceSynchronize());

    // allocate FB
    vec3 *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    dim3 blocks(nx/tx+1,ny/ty+1);
    dim3 threads(tx,ty);

    float v_height = 5.0;
    float v_width = nx/ny * v_height;
    float fov = 45;
    float focal_length = (v_width/2) / tan(DEG_TO_RAD(fov/2));

    config_t config = {vec3(0, 0, 0), nx, ny, focal_length, v_height};

    render<<<blocks, threads>>>(fb, config, scene, n_objs);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    /* Output as PPM Image */
    std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny-1; j >= 0; j--) {
        std::clog << "\rScanlines remaining: " << (ny - j) << ' ' << std::flush;
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j*nx + i;
            float r = fb[pixel_index].x();
            float g = fb[pixel_index].y();
            float b = fb[pixel_index].z();
            int ir = int(255.99*r);
            int ig = int(255.99*g);
            int ib = int(255.99*b);
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }

    checkCudaErrors(hipFree(fb));
    checkCudaErrors(hipFree(scene));
}