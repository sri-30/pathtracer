#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>

//#include "vec3.h"
#include "color.h"
#include "shapes.h"
#include "util.h"
#include <math.h>
#include <time.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define MATT_RED Material(color3(0.4, 0, 0), color3(0.4, 0, 0))
#define MATT_GREEN Material(color3(0, 0.4, 0), color3(0, 0.4, 0))
#define MATT_BLUE Material(color3(0, 0, 0.4), color3(0, 0, 0.4))
#define MATT_WHITE Material(color3(0.5, 0.5, 0.5), color3(0.5, 0.5, 0.5))

#define EPSILON 0.0001


#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

typedef struct config_s {
    vec3 camera_pos;
    int max_x;
    int max_y;
    float focal_length;
    float view_port_height;
} config_t;

__device__ IntersectionPoint getNearestIntersection(ray& r, Shape** scene, int n_objects) {
    IntersectionPoint min_p;
    min_p.intersects = false;
    for (int k = 0; k < n_objects; k++) {
        Shape *obj = scene[k];
        RayPath p = obj->getIntersections(r);
        IntersectionPoint point;
        if (p.n_intersections == 1) {
            point = p.first;
        } else if (p.n_intersections == 2) {
            point = (p.first.distance < p.second.distance) ? p.first : p.second;
        } else {
            point.intersects = false;
        }
        if (point.intersects && !min_p.intersects || point.distance < min_p.distance) {
            min_p = point;
            if (min_p.normal.dot(r.direction()) > 0){
                min_p.normal = -1 * min_p.normal;}
        }
    }
    return min_p;
}



__device__ vec3 randomDirectionHemisphere(vec3 normal, hiprandState *s) {
    vec3 res(hiprand_normal(s), hiprand_normal(s), hiprand_normal(s));
    res.normalize();
    normal.normalize();
    res = res.dot(normal) < 0 ? -1 * res : res;
    return res.normalized();
}

__device__ vec3 cosineWeightedRandomDirectionHemisphere(vec3 normal, hiprandState *s) {
    float rv1 = hiprand_normal(s);
    float rv2 = hiprand_normal(s);
    
	vec3  uu = normal.cross(vec3(0.0,1.0,1.0)).normalized();
	vec3  vv = uu.cross(normal).normalized();
	
	float ra = sqrt(rv2);
	float rx = ra*cos(6.2831*rv1); 
	float ry = ra*sin(6.2831*rv1);
	float rz = sqrt( 1.0-rv2 );
	vec3  rr = vec3( rx*uu + ry*vv + rz*normal );
    
    return rr.normalized();
}

__device__ vec3 tracePath(Shape **scene, int n_objects, ray& r, hiprandState *s) {
    vec3 contribution(0, 0, 0);
    vec3 tp(1, 1, 1);
    int n_bounces = 50;
    vec3 coefficient = vec3(1, 1, 1);
    int refractive_index_current = 1;

    for (int i = 0; i <= n_bounces; i++) {
        IntersectionPoint min_p = getNearestIntersection(r, scene, n_objects);
        if (!min_p.intersects)
            break;
        Material material = min_p.material;
        if (material.refractive) {
            vec3 refract_dir = refract(r.direction(), min_p.normal, refractive_index_current, (refractive_index_current == 1 ? material.refractive_index : 1));
            //printf("A: %9.6f B: %9.6f C: %0.6f\n", refract_dir[0], refract_dir[1], refract_dir[2]);
            if (refract_dir.norm() != 0){
                r = ray(min_p.position, refract_dir.normalized());
                n_bounces++;
                refractive_index_current = (refractive_index_current == 1 ? material.refractive_index : 1);
            } else {
                r = ray(min_p.position, reflect(r.direction(), min_p.normal).normalized());
                n_bounces++;
            }
            r = ray(r.origin() + EPSILON * r.direction(), r.direction());
            continue;
        }
        color3 emittance = material.color_emission * material.emissive;
        vec3 newOrigin = min_p.position;
        vec3 newDirection = randomDirectionHemisphere(min_p.normal, s);
        const float p = 1 / (2 * PI);
        float cos_theta = newDirection.dot(min_p.normal);
        color3 brdf = material.color_reflection / PI;
        contribution += coefficient.cwiseProduct(emittance);
        coefficient = coefficient.cwiseProduct(brdf * (cos_theta / p));
        r = ray(newOrigin, newDirection);
    }
    return contribution;
}

__global__ void render(vec3 *fb, config_t config, Shape** scene, int n_objects, LightSource** lights, int n_lights) {
    int max_x = config.max_x;
    int max_y = config.max_y;
    
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x || j >= max_y))
        return;
    // if ((i <= 5) && (j < 105 || j > 110))
    //     return;
    int pixel_index = j*max_x + i;

    vec3 camera_pos = config.camera_pos;
    float focal_length = config.focal_length;
    float viewport_height = config.view_port_height;
    float viewport_width = viewport_height * (static_cast<float>(max_x)/max_y);
    
    vec3 viewport_u = vec3(viewport_width, 0, 0);
    vec3 viewport_v = vec3(0, -viewport_height, 0);

    vec3 pixel_delta_u = viewport_u / (float) max_x;
    vec3 pixel_delta_v = viewport_v / (float) max_y;

    vec3 viewport_upper_left = camera_pos - vec3(0, 0, focal_length) - viewport_u/2.0 - viewport_v/2.0;
    vec3 pixel00_loc = viewport_upper_left + (pixel_delta_u + pixel_delta_v) * 0.5;

    //uint32_t rngState = ((uint32_t)(i) * (uint32_t)(1973) + (uint32_t)(j) * (uint32_t) (9277) ) | uint(1);


    color3 ambient(0.3, 0.3, 0.3);
    int n_samples = 100;
    color3 p_color(0, 0, 0);
    color3 rayColor(1, 1, 1);
    color3 incomingLight(0, 0, 0);
    color3 totalLight(0, 0, 0);
    int n_origin;
    hiprandState s;
    // printf("A: %9.6f B: %9.6f C: %0.6f\n", direction[0], direction[1], direction[2]);
    // printf("A1: %9.6f B1: %9.6f C1: %0.6f\n", pixel_delta_v[0], pixel_delta_v[1], pixel_delta_v[2]);
    hiprand_init(pixel_index, 0, 0, &s);
    for (int sample = 0; sample < n_samples; sample++) {
        vec3 p_center = pixel00_loc + (pixel_delta_u * (j + 0 * hiprand_uniform(&s))) + (pixel_delta_v * (i + 0 * hiprand_uniform(&s)));
        vec3 direction = (p_center - camera_pos).normalized();
        ray r = ray(camera_pos, direction);
        totalLight += tracePath(scene, n_objects, r, &s);
    }
    p_color = totalLight/n_samples;
    fb[pixel_index] = p_color;
}

__global__ void constructScene(Shape **scene) {
    if (threadIdx.x == 0 && blockIdx.x == 0){
        Eigen::Affine3f t1 = IDENTITY;
        t1.translation() = Eigen::Translation3f(0.4, 0, -4.0).translation();
        
        /* Floor */
        Eigen::Affine3f tplane1 = IDENTITY;
        tplane1.linear() = Eigen::AngleAxisf(PI/2, vec3(0, 1, 0)).toRotationMatrix();
        tplane1.translation() = Eigen::Translation3f(-1.0, 0, 0.0).translation();

        /* Back Wall */
        Eigen::Affine3f tplane2 = IDENTITY;
        //t2.linear() = Eigen::AngleAxisf(PI/2, vec3(0, 1, 0)).toRotationMatrix();
        tplane2.translation() = Eigen::Translation3f(0.0, 0, -5.0).translation();

        /* Left Wall */
        Eigen::Affine3f tplane3 = IDENTITY;
        tplane3.linear() = Eigen::AngleAxisf(PI/2, vec3(1, 0, 0)).toRotationMatrix();
        tplane3.translation() = Eigen::Translation3f(0.0, -1.0, -5.0).translation();
        
        /* Right Wall */
        Eigen::Affine3f tplane4 = IDENTITY;
        tplane4.linear() = Eigen::AngleAxisf(-PI/2, vec3(1, 0, 0)).toRotationMatrix();
        tplane4.translation() = Eigen::Translation3f(0.0, 1.0, -5.0).translation();
                
        /* Ceiling */
        Eigen::Affine3f tplane5 = IDENTITY;
        tplane5.linear() = Eigen::AngleAxisf(-PI/2, vec3(0, 1, 0)).toRotationMatrix();
        tplane5.translation() = Eigen::Translation3f(1.5, 0.0, 0.0).translation();

        Eigen::Affine3f t3 = IDENTITY;
        t3.translation() = Eigen::Translation3f(-0.5, 0.5, -4.0).translation();

        Eigen::Affine3f t4 = IDENTITY;
        t4.translation() = Eigen::Translation3f(-0.5, -0.5, -3.0).translation();

        Eigen::Affine3f t5 = IDENTITY;
        t5.translation() = Eigen::Translation3f(-0.5, 0.5, -3.0).translation();

        Material light_material;
        light_material.color_reflection = color3(0, 0, 0);
        light_material.color_emission = color3(1, 1, 1);
        light_material.emissive = 20;

        Material glass;
        glass.refractive = true;
        glass.refractive_index = 1.5f;
        
        //scene[0] = new Cylinder(MATT_RED, t2);
        
        scene[0] = new Plane(MATT_WHITE, tplane1, -5, -5, 15, 15);
        scene[1] = new Plane(MATT_WHITE, tplane2, -5, -5, 15, 15);
        scene[2] = new Plane(MATT_BLUE, tplane3, -5, -5, 15, 15);
        scene[3] = new Plane(MATT_BLUE, tplane4, -5, -5, 15, 15);
        scene[4] = new Plane(MATT_WHITE, tplane5, -5, -5, 15, 15);
        
        scene[5] = new Sphere(light_material, t1);
        scene[6] = new Sphere(MATT_RED, t4);
        scene[7] = new Sphere(MATT_GREEN, t3);
        //scene[8] = new Sphere(glass, t5);
    }
}

// __global__ void constructTransforms(Eigen::Affine3f **transforms) {
//         Eigen::Affine3f a = IDENTITY;
//         a.linear() = Eigen::AngleAxisf(10, vec3(0, 0, 1)).toRotationMatrix() * Eigen::Matrix3f::Identity();
//         a.translation() = Eigen::Translation3f(0, 0, -5.0).translation();
// }


__global__ void constructLights(LightSource **lights) {
    if (threadIdx.x == 0 && blockIdx.x == 0)
        lights[0] = new LightSource(vec3(0.5, 0.5, 0), color3(1, 1, 1), 120);
}


int main() {

    int nx = 400;
    int ny = 400;

    int tx = 8;
    int ty = 8;

    int num_pixels = nx*ny;

    size_t fb_size = num_pixels*sizeof(color3);

    // allocate space for scene
    int n_objs = 8;
    Shape **scene;

    int n_lights = 1;
    LightSource **lights;

    Eigen::Affine3f **transforms;
    checkCudaErrors(hipMalloc((void **)&scene, n_objs*sizeof(void**)));
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipMalloc((void **)&lights, n_lights*sizeof(void**)));
    checkCudaErrors(hipDeviceSynchronize());
    // checkCudaErrors(hipMalloc((void **)&transforms, n_objs*sizeof(void**)));
    // checkCudaErrors(hipDeviceSynchronize());
    constructScene<<<1, 1>>>(scene);
    checkCudaErrors(hipDeviceSynchronize());
    //constructTransforms<<<1, 1>>>(transforms);
    constructLights<<<1, 1>>>(lights);
    checkCudaErrors(hipDeviceSynchronize());

    // allocate FB
    color3 *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    dim3 blocks(nx/tx+1,ny/ty+1);
    dim3 threads(tx,ty);

    float v_height = 5.0;
    float v_width = nx/ny * v_height;
    float fov = 45;
    float focal_length = (v_width/2) / tan(DEG_TO_RAD(fov/2));

    config_t config = {vec3(0, 0, 0), nx, ny, focal_length, v_height};

    hiprandState *devStates;
    //checkCudaErrors(hipMalloc((void **)&devStates, blocks * threads *sizeof(hiprandState)));

    render<<<blocks, threads>>>(fb, config, scene, n_objs, lights, n_lights);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    // Output FB as Image
    std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny-1; j >= 0; j--) {
        std::clog << "\rScanlines remaining: " << (ny - j) << ' ' << std::flush;
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j*nx + i;
            float r = fb[pixel_index].x();
            float g = fb[pixel_index].y();
            float b = fb[pixel_index].z();
            int ir = int(255.99*r);
            int ig = int(255.99*g);
            int ib = int(255.99*b);
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }

    checkCudaErrors(hipFree(fb));
    checkCudaErrors(hipFree(scene));
    checkCudaErrors(hipFree(lights));
}