#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>

//#include "vec3.h"
#include "color.h"
#include "shapes.h"
#include "util.h"
#include <math.h>
#include <time.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define MATT_RED Material(color3(0.4, 0, 0), color3(0.4, 0, 0))
#define MATT_GREEN Material(color3(0, 0.4, 0), color3(0, 0.4, 0))
#define MATT_BLUE Material(color3(0, 0, 0.4), color3(0, 0, 0.4))
#define MATT_WHITE Material(color3(0.5, 0.5, 0.5), color3(0.5, 0.5, 0.5))

#define EPSILON 0.0001


#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__device__ float lerp(float v0, float v1, float t) {
  return (1 - t) * v0 + t * v1;
}

__device__ vec3 lerp_vec(vec3 v0, vec3 v1, float t) {
  return (1 - t) * v0 + t * v1;
}


typedef struct config_s {
    vec3 camera_pos;
    int max_x;
    int max_y;
    float focal_length;
    float view_port_height;
} config_t;

__device__ IntersectionPoint getNearestIntersection(ray& r, Shape** scene, int n_objects) {
    IntersectionPoint min_p;
    min_p.intersects = false;
    for (int k = 0; k < n_objects; k++) {
        Shape *obj = scene[k];
        RayPath p = obj->getIntersections(r);
        IntersectionPoint point;
        if (p.n_intersections == 1) {
            point = p.first;
        } else if (p.n_intersections == 2) {
            point = (p.second.distance < p.first.distance) ? p.second : p.first;
        } else {
            point.intersects = false;
        }
        if (point.intersects && (!min_p.intersects || point.distance < min_p.distance)) {
            min_p = point;
            min_p.inside = min_p.normal.dot(r.direction()) > 0;
            if (min_p.inside) {min_p.normal = -1 * min_p.normal;}
        }
    }
    return min_p;
}



__device__ vec3 randomDirectionHemisphere(vec3 normal, hiprandState *s) {
    vec3 res(hiprand_normal(s), hiprand_normal(s), hiprand_normal(s));
    res.normalize();
    normal.normalize();
    res = res.dot(normal) < 0 ? -1 * res : res;
    return res.normalized();
}

__device__ vec3 cosineWeightedRandomDirectionHemisphere(vec3 normal, hiprandState *s) {
    float rv1 = hiprand_normal(s);
    float rv2 = hiprand_normal(s);
    
	vec3  uu = normal.cross(vec3(0.0,1.0,1.0)).normalized();
	vec3  vv = uu.cross(normal).normalized();
	
	float ra = sqrt(rv2);
	float rx = ra*cos(6.2831*rv1); 
	float ry = ra*sin(6.2831*rv1);
	float rz = sqrt( 1.0-rv2 );
	vec3  rr = vec3( rx*uu + ry*vv + rz*normal );
    
    return rr.normalized();
}

// __device__ vec3 sampleLights(LightSource **lights, int n_lights, vec3 position) {
//     vec3 res(0, 0, 0);
//     for (int i = 0; i < n_lights; i++) {
//         res += lights[i]->emission_color*(lights[i]->getIntensity((lights[i]->position - position).norm()));
//     }
//     return res;
// } 

__device__ Eigen::Quaternionf getRotationToZAxis(vec3 input) {

	// Handle special case when input is exact or near opposite of (0, 0, 1)
	if (input.z() < -0.99999f) return Eigen::Quaternionf(1.0f, 0.0f, 0.0f, 0.0f);

	return Eigen::Quaternionf(1.0f + input.z(), input.y(), -input.x(), 0.0f).normalized();
}


__device__ Eigen::Quaternionf getRotationFromZAxis(vec3 input) {

	// Handle special case when input is exact or near opposite of (0, 0, 1)
	if (input.z() < -0.99999f) return Eigen::Quaternionf(1.0f, 0.0f, 0.0f, 0.0f);

	return Eigen::Quaternionf(1.0f + input.z(), input.y(), input.x(), 0.0f).normalized();
}

__device__ Eigen::Quaternionf invertRotation(Eigen::Quaternionf q) {
    return Eigen::Quaternionf(q.w(), -q.x(), -q.y(), -q.z());
}

/* Cosine-Weighted Distribution Sampling */
__device__ vec3 sampleHemisphere(float u1, float u2) {
    // float a = sqrt(x);
    // float b = 2*PI*y;

    // return vec3(a*cos(b), a*sin(b), sqrt(1 - x));
    float z = u1 * 2.0f - 1.0f;
    float a = u2 * 2*PI;
    float r = sqrt(1.0f - z * z);
    float x = r * cos(a);
    float y = r * sin(a);
    return vec3(x, y, z);
}

__device__ vec3 sampleGGXWalter(vec3 Vlocal, float alpha, float x, float y) {
	float alphaSquared = pow(alpha, 2);

	// Calculate cosTheta and sinTheta needed for conversion to H vector
	float cosThetaSquared = (1.0f - x) / ((alphaSquared - 1.0f) * x + 1.0f);
	float cosTheta = sqrt(cosThetaSquared);
	float sinTheta = sqrt(1.0f - cosThetaSquared);
	float phi = 2 * PI * y;

	// Convert sampled spherical coordinates to H vector
	return vec3(sinTheta * cos(phi), sinTheta * sin(phi), cosTheta).normalized();
}

__device__ vec3 evalFresnelSchlick(vec3 f0, float NdotS)
{
	return f0 + (vec3(1, 1, 1) - f0) * pow(1.0f - NdotS, 5.0f);
}

__device__ float luminance(vec3 rgb)
{
	return rgb.dot(vec3(0.2126f, 0.7152f, 0.0722f));
}

__device__ float FresnelReflectAmount(float n1, float n2, vec3 normal, vec3 incident, float f0, float f90)
{
        // Schlick aproximation
        float r0 = (n1-n2) / (n1+n2);
        r0 *= r0;
        float cosX = -normal.dot(incident);
        if (n1 > n2)
        {
            float n = n1/n2;
            float sinT2 = n*n*(1.0-cosX*cosX);
            // Total internal reflection
            if (sinT2 > 1.0)
                return f90;
            cosX = sqrt(1.0-sinT2);
        }
        float x = 1.0-cosX;
        float ret = r0+(1.0-r0)*x*x*x*x*x;

        // adjust reflect multiplier for object reflectivity
        return lerp(f0, f90, ret);
}


__device__ vec3 tracePath(Shape **scene, int n_objects, ray& r, hiprandState *s) {
    vec3 contribution(0.0f, 0.0f, 0.0f);
    int n_bounces = 100;
    vec3 coefficient = vec3(1.0f, 1.0f, 1.0f);

    for (int i = 0; i <= n_bounces; i++) {
        // shoot a ray out into the world
        IntersectionPoint min_p = getNearestIntersection(r, scene, n_objects);
        
        // if the ray missed, we are done
        if (!min_p.intersects){
            break;
        }

        Material material = min_p.material;
        
        // do absorption if we are hitting from inside the object
        if (i > 0 && min_p.inside) {
            vec3 attenuation = -1 * min_p.material.refractionColor * min_p.distance;
            coefficient = coefficient.cwiseProduct(vec3(exp(attenuation[0]), exp(attenuation[1]), exp(attenuation[2])));
        }
        
        // get the pre-fresnel chances
        float specularChance = material.specularChance;
        float refractionChance = material.refractionChance;
        
        // take fresnel into account for specularChance and adjust other chances.
        // specular takes priority.
        // chanceMultiplier makes sure we keep diffuse / refraction ratio the same.
        float rayProbability = 1.0f;
        if (specularChance > 0.0f)
        {
            specularChance = FresnelReflectAmount(
            	min_p.inside ? material.IOR : 1.0,
            	!min_p.inside ? material.IOR : 1.0,
            	r.direction(), min_p.normal, min_p.material.specularChance, 1.0f);
            
            float chanceMultiplier = (1.0f - specularChance) / (1.0f - min_p.material.specularChance);
            refractionChance *= chanceMultiplier;
        }
        
        // calculate whether we are going to do a diffuse, specular, or refractive ray
        float doSpecular = 0.0f;
        float doRefraction = 0.0f;
        float raySelectRoll = hiprand_uniform(s);
		if (specularChance > 0.0f && raySelectRoll < specularChance)
        {
            doSpecular = 1.0f;
            rayProbability = specularChance;
        }
        else if (refractionChance > 0.0f && raySelectRoll < specularChance + refractionChance)
        {
            doRefraction = 1.0f;
            rayProbability = refractionChance;
        }
        else
        {
            rayProbability = 1.0f - (specularChance + refractionChance);
        }

        // if (doRefraction != 1.0f) {
        //     printf("yes\n");
        // }
        
        // numerical problems can cause rayProbability to become small enough to cause a divide by zero.
		rayProbability = max(rayProbability, 0.001f);

        vec3 newPosition = min_p.position;
        // Calculate a new ray direction.
        // Diffuse uses a normal oriented cosine weighted hemisphere sample.
        // Perfectly smooth specular uses the reflection ray.
        // Rough (glossy) specular lerps from the smooth specular to the rough diffuse by the material roughness squared
        // Squaring the roughness is just a convention to make roughness feel more linear perceptually.
        vec3 diffuseRayDir = (min_p.normal + sampleHemisphere(hiprand_uniform(s), hiprand_uniform(s))).normalized();
        
        vec3 specularRayDir = reflect(r.direction(), min_p.normal).normalized();
        specularRayDir = (lerp_vec(specularRayDir, diffuseRayDir, material.specularRoughness*material.specularRoughness)).normalized();

        vec3 refractionRayDir = refract(r.direction(), min_p.normal, min_p.inside ? min_p.material.IOR : 1.0f / min_p.material.IOR).normalized();
        
        refractionRayDir = (lerp_vec(refractionRayDir, (min_p.normal + sampleHemisphere(hiprand_uniform(s), hiprand_uniform(s))).normalized(), material.refractionRoughness*material.refractionRoughness).normalized());
        
        vec3 newDirection = lerp_vec(diffuseRayDir, specularRayDir, doSpecular).normalized();
        newDirection = lerp_vec(newDirection, refractionRayDir, doRefraction).normalized();
        
		// add in emissive lighting
        if (material.emissive.norm() > 0){
            contribution = contribution + material.emissive.cwiseProduct(coefficient);
        }
        
        // update the colorMultiplier. refraction doesn't alter the color until we hit the next thing, so we can do light absorption over distance.
        if (doRefraction == 0.0f)
            coefficient = coefficient.cwiseProduct(lerp_vec(material.albedo, material.specularColor, doSpecular));
        
        // since we chose randomly between diffuse, specular, refract,
        // we need to account for the times we didn't do one or the other.
        coefficient = coefficient / rayProbability;
        
        // Russian Roulette
        // As the throughput gets smaller, the ray is more likely to get terminated early.
        // Survivors have their value boosted to make up for fewer samples being in the average.
        float p = max(coefficient.x(), max(coefficient.y(), coefficient.z()));
        if (hiprand_uniform(s) > p)
            break;

        // Add the energy we 'lose' by randomly terminating paths
        coefficient = coefficient * 1.0f / p;    
        r = ray(newPosition, newDirection); 
    }
    return contribution;
}

__global__ void render(vec3 *fb, config_t config, Shape** scene, int n_objects, LightSource** lights, int n_lights) {
    int max_x = config.max_x;
    int max_y = config.max_y;
    
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x || j >= max_y))
        return;
    // if ((i < 200 || i > 205) || (j < 200 || j > 205))
    //     return;
    int pixel_index = j*max_x + i;

    vec3 camera_pos = config.camera_pos;
    float focal_length = config.focal_length;
    float viewport_height = config.view_port_height;
    float viewport_width = viewport_height * (static_cast<float>(max_x)/max_y);
    
    vec3 viewport_u = vec3(viewport_width, 0, 0);
    vec3 viewport_v = vec3(0, -viewport_height, 0);

    vec3 pixel_delta_u = viewport_u / (float) max_x;
    vec3 pixel_delta_v = viewport_v / (float) max_y;

    vec3 viewport_upper_left = camera_pos - vec3(0, 0, focal_length) - viewport_u/2.0 - viewport_v/2.0;
    vec3 pixel00_loc = viewport_upper_left + (pixel_delta_u + pixel_delta_v) * 0.5;

    //uint32_t rngState = ((uint32_t)(i) * (uint32_t)(1973) + (uint32_t)(j) * (uint32_t) (9277) ) | uint(1);


    color3 ambient(0.3, 0.3, 0.3);
    int n_samples = 2000;
    color3 p_color(0, 0, 0);
    color3 rayColor(1, 1, 1);
    color3 incomingLight(0, 0, 0);
    color3 totalLight(0, 0, 0);
    int n_origin;
    hiprandState s;
    // printf("A: %9.6f B: %9.6f C: %0.6f\n", direction[0], direction[1], direction[2]);
    // printf("A1: %9.6f B1: %9.6f C1: %0.6f\n", pixel_delta_v[0], pixel_delta_v[1], pixel_delta_v[2]);
    hiprand_init(pixel_index, 0, 0, &s);
    for (int sample = 0; sample < n_samples; sample++) {
        vec3 p_center = pixel00_loc + (pixel_delta_u * (j - 0.5f + hiprand_uniform(&s))) + (pixel_delta_v * (i - 0.5f + hiprand_uniform(&s)));
        vec3 direction = (p_center - camera_pos).normalized();
        ray r = ray(camera_pos, direction);
        totalLight += tracePath(scene, n_objects, r, &s);
    }
    p_color = totalLight/((float)n_samples);
    fb[pixel_index] = p_color;
}

__global__ void constructScene(Shape **scene) {
    if (threadIdx.x == 0 && blockIdx.x == 0){
        Eigen::Affine3f t1 = IDENTITY;
        t1.translation() = Eigen::Translation3f(0.5, 0, -3.0).translation();
        
        /* Floor */
        Eigen::Affine3f tplane1 = IDENTITY;
        tplane1.linear() = Eigen::AngleAxisf(PI/2, vec3(0, 1, 0)).toRotationMatrix();
        tplane1.translation() = Eigen::Translation3f(-1.0, 0, 0.0).translation();

        /* Back Wall */
        Eigen::Affine3f tplane2 = IDENTITY;
        //t2.linear() = Eigen::AngleAxisf(PI/2, vec3(0, 1, 0)).toRotationMatrix();
        tplane2.translation() = Eigen::Translation3f(0.0, 0, -5.0).translation();

        /* Left Wall */
        Eigen::Affine3f tplane3 = IDENTITY;
        tplane3.linear() = Eigen::AngleAxisf(PI/2, vec3(1, 0, 0)).toRotationMatrix();
        tplane3.translation() = Eigen::Translation3f(0.0, -1.0, -5.0).translation();
        
        /* Right Wall */
        Eigen::Affine3f tplane4 = IDENTITY;
        tplane4.linear() = Eigen::AngleAxisf(-PI/2, vec3(1, 0, 0)).toRotationMatrix();
        tplane4.translation() = Eigen::Translation3f(0.0, 1.0, -5.0).translation();
                
        /* Ceiling */
        Eigen::Affine3f tplane5 = IDENTITY;
        tplane5.linear() = Eigen::AngleAxisf(-PI/2, vec3(0, 1, 0)).toRotationMatrix();
        tplane5.translation() = Eigen::Translation3f(1.5, 0.0, 0.0).translation();

        Eigen::Affine3f t3 = IDENTITY;
        t3.translation() = Eigen::Translation3f(-0.5, 0.5, -4.0).translation();

        Eigen::Affine3f t4 = IDENTITY;
        //t4.linear() = Eigen::AngleAxisf(PI/4, vec3(1, 0, 0)).toRotationMatrix();
        t4.translation() = Eigen::Translation3f(-0.5, -0.5, -3.0).translation();

        Eigen::Affine3f t5 = IDENTITY;
        t5.translation() = Eigen::Translation3f(0.0, 0.0, -2.0).translation();
        t5.linear() = Eigen::AngleAxisf(PI/4, vec3(1, 0, 0)).toRotationMatrix();

        Material light_material;
        light_material.emissive = vec3(1.0f, 0.9f, 0.7f) * 10.0f;   
        
        //scene[0] = new Cylinder(MATT_RED, t2);

        Material base;
        base.albedo = vec3(0.9f, 0.25f, 0.25f);
        base.emissive = vec3(0.0f, 0.0f, 0.0f);        
        base.specularChance = 0.0f;
        base.specularRoughness = 0.0f;
        base.specularColor = vec3(1.0f, 1.0f, 1.0f) * 0.8f;
        base.IOR = 1.1f;
        base.refractionChance = 0.0f;

        Material blue_wall = base;
        blue_wall.IOR = 1.0f;
        Material white_wall = base;
        white_wall.IOR = 1.0f;
        Material metallic_ball = base;
        Material dielectric_ball = base;
        dielectric_ball.IOR = 1.0f;
        Material white_ball = base;
        Material floor = base;

        Material glass;
        glass.albedo = vec3(0.9f, 0.25f, 0.25f);
        glass.emissive = vec3(0.0f, 0.0f, 0.0f);        
        glass.specularChance = 0.02f;
        glass.specularRoughness = 0;
        glass.specularColor = vec3(1.0f, 1.0f, 1.0f) * 0.8f;
        glass.IOR = 1.1f;
        glass.refractionChance = 1.0f;
        glass.refractionRoughness = 0;
        glass.refractionColor = vec3(0.0f, 0.5f, 1.0f);

        floor.albedo = vec3(0.4, 0.01, 0.3);

        metallic_ball.specularChance = 0.3;

        blue_wall.albedo = vec3(0, 0, 0.4f);
        white_wall.albedo = vec3(0.2f, 0.2f, 0.2f);
        metallic_ball.albedo = vec3(0.4f, 0, 0);
        dielectric_ball.albedo = vec3(0, 0.4f, 0);
        white_ball.albedo = vec3(0.9f, 0.25f, 0.25f);

        scene[0] = new Plane(white_wall, tplane1, -5, -5, 15, 15);
        scene[1] = new Plane(white_wall, tplane2, -5, -5, 15, 15);
        scene[2] = new Plane(white_wall, tplane3, -5, -5, 15, 15);
        scene[3] = new Plane(white_wall, tplane4, -5, -5, 15, 15);
        scene[4] = new Plane(light_material, tplane5, -5, -5, 15, 15);
        
        scene[5] = new Sphere(dielectric_ball, t3);
        scene[6] = new Sphere(metallic_ball, t4);
        //scene[7] = new Sphere(dielectric_ball, t3);
        //scene[8] = new Cube(glass, t5);
    }
}

// __global__ void constructTransforms(Eigen::Affine3f **transforms) {
//         Eigen::Affine3f a = IDENTITY;
//         a.linear() = Eigen::AngleAxisf(10, vec3(0, 0, 1)).toRotationMatrix() * Eigen::Matrix3f::Identity();
//         a.translation() = Eigen::Translation3f(0, 0, -5.0).translation();
// }


__global__ void constructLights(LightSource **lights) {
    if (threadIdx.x == 0 && blockIdx.x == 0)
        lights[0] = new LightSource(vec3(0.5, 0.5, 0), color3(1, 1, 1), 120);
}


int main() {

    int nx = 400;
    int ny = 400;

    int tx = 8;
    int ty = 8;

    int num_pixels = nx*ny;

    size_t fb_size = num_pixels*sizeof(color3);

    // allocate space for scene
    int n_objs = 7;
    Shape **scene;

    int n_lights = 1;
    LightSource **lights;

    checkCudaErrors(hipMalloc((void **)&scene, n_objs*sizeof(void**)));
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipMalloc((void **)&lights, n_lights*sizeof(void**)));
    checkCudaErrors(hipDeviceSynchronize());
    constructScene<<<1, 1>>>(scene);
    checkCudaErrors(hipDeviceSynchronize());
    constructLights<<<1, 1>>>(lights);
    checkCudaErrors(hipDeviceSynchronize());

    // allocate FB
    color3 *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    dim3 blocks(nx/tx+1,ny/ty+1);
    dim3 threads(tx,ty);

    float v_height = 5.0;
    float v_width = nx/ny * v_height;
    float fov = 45;
    float focal_length = (v_width/2) / tan(DEG_TO_RAD(fov/2));

    config_t config = {vec3(0, 0, 0), nx, ny, focal_length, v_height};

    render<<<blocks, threads>>>(fb, config, scene, n_objs, lights, n_lights);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    // Output FB as Image
    std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny-1; j >= 0; j--) {
        std::clog << "\rScanlines remaining: " << (ny - j) << ' ' << std::flush;
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j*nx + i;
            float r = fb[pixel_index].x();
            float g = fb[pixel_index].y();
            float b = fb[pixel_index].z();
            int ir = int(255.99*r);
            int ig = int(255.99*g);
            int ib = int(255.99*b);
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }

    checkCudaErrors(hipFree(fb));
    checkCudaErrors(hipFree(scene));
    checkCudaErrors(hipFree(lights));
}